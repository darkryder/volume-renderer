#include "hip/hip_runtime.h"
#include "device_helpers.cuh"

rtDeclareVariable(float3,   eye, , );
rtDeclareVariable(float3,   U, , );
rtDeclareVariable(float3,   V, , );
rtDeclareVariable(float3,   W, , );

rtDeclareVariable(uint2,    launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2,    launch_dim, rtLaunchDim, );

rtBuffer<uchar4, 2>         output_buffer;

RT_PROGRAM void pinhole_camera() {
    float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
    float3 ray_origin = eye;
    float3 ray_direction = optix::normalize(d.x*U + d.y*V + W);

    output_buffer[launch_index] = make_colour(
        // optix::make_float3(1.0f, 1.0f, 1.0f)
        ray_direction*2
    );
}
