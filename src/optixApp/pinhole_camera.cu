#include "hip/hip_runtime.h"
#include "device_helpers.cuh"

rtDeclareVariable(float3,   eye, , );
rtDeclareVariable(float3,   U, , );
rtDeclareVariable(float3,   V, , );
rtDeclareVariable(float3,   W, , );

rtDeclareVariable(uint,     volume_width, ,);
rtDeclareVariable(uint,     volume_height, ,);
rtDeclareVariable(uint,     volume_depth, ,);

rtDeclareVariable(uint2,    launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2,    launch_dim, rtLaunchDim, );

rtDeclareVariable(rtObject, top_object, ,);

rtBuffer<uchar4, 2>                 output_buffer;

RT_PROGRAM void pinhole_camera() {
    float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
    float3 ray_origin = eye;
    float3 ray_direction = optix::normalize(d.x*U + d.y*V + W);

/*#ifdef DEBUG
    float t = tex3D(
        volume_texture,
        launch_index.x/(float) volume_width,
        launch_index.y/(float) volume_height,
        (frame_idx % volume_depth)/(float) volume_depth
    );
    if (launch_index.x == 128 && launch_index.y == 128) rtPrintf("Setting t to %f for %d %d %d \n", t, launch_index.x, launch_index.y, frame_idx);
#else
    float t = tex3D(volume_texture, launch_index.x, launch_index.y, 0);
    if (launch_index.x == 64 && launch_index.y == 64) rtPrintf("Setting t to %f for %d %d %d\n", t, launch_index.x, launch_index.y, frame_idx);
#endif

*/
    optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, 0.f, RT_DEFAULT_MAX);
    PerRayData_radiance prd;
    prd.result = 0;

    rtTrace(top_object, ray, prd);

    float3 transferred = transfer_function(prd.result);

    // if (launch_index.x == 0 && launch_index.y == 0)
    // rtPrintf("%f\n\n", transferred);

    output_buffer[launch_index] = make_colour(
        // optix::make_float3(1.0f, 1.0f, 1.0f)
        // optix::make_float3(t/255.0f, t/255.0f, t/255.0f)
        // ray_direction*2
        transferred
    );
}
