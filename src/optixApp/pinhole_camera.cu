#include "hip/hip_runtime.h"
#include "device_helpers.cuh"

rtDeclareVariable(float3,   eye, , );
rtDeclareVariable(float3,   U, , );
rtDeclareVariable(float3,   V, , );
rtDeclareVariable(float3,   W, , );
rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(uint2,    launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2,    launch_dim, rtLaunchDim, );

rtBuffer<uchar4, 2>         output_buffer;

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth;
};

RT_PROGRAM void pinhole_camera() {
    float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
    float3 ray_origin = eye;
    float3 ray_direction = optix::normalize(d.x*U + d.y*V + W);

    optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0, 0.f, RT_DEFAULT_MAX);

    PerRayData_radiance prd;
    prd.importance = 1.f;
    prd.depth = 0;

    rtTrace(top_object, ray, prd);

    output_buffer[launch_index] = make_colour(
        // optix::make_float3(1.0f, 1.0f, 1.0f)
        ray_direction*2
    );
}
