#include "hip/hip_runtime.h"
#include "device_helpers.cuh"

rtBuffer<float, 3>         volume_data;
rtDeclareVariable(uint,    volume_width, , );
rtDeclareVariable(uint,    volume_height, , );

// http://math.stackexchange.com/a/19780
RT_PROGRAM void bounding_box(int prim_idx, float result[6]) {
    optix::Aabb *aabb = (optix::Aabb *) result;
    uint voxel_k = prim_idx / (volume_width*volume_height);
    uint voxel_j = (prim_idx - voxel_k*volume_width*volume_height) / volume_width;
    uint voxel_i = prim_idx - voxel_k*volume_width*volume_height - voxel_j*volume_width;

    aabb->m_min = optix::make_float3(voxel_i, voxel_j, voxel_k);
    aabb->m_max = optix::make_float3(voxel_i + 1, voxel_j + 1, voxel_k + 1);
}
