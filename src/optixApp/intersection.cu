#include "hip/hip_runtime.h"
#include "device_helpers.cuh"
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(uint2,    launch_index, rtLaunchIndex, );

rtDeclareVariable(uint, volume_width, ,);
rtDeclareVariable(uint, volume_height, ,);
rtDeclareVariable(uint, volume_depth, ,);

rtDeclareVariable(float, stepping_distance, ,);

rtTextureSampler<float, 3>  volume_texture;
rtTextureSampler<float4, 1> transfer_fn_texture;

#define MAX_STEPS 50000
#define kassert( X ) if ( !(X) ) {\
 return ;}

#define EPS 5.0f


static __device__ bool get_aabb_ray_intersection(float &tmin, float &tmax) {
    optix::float3 orig = ray.origin;
    optix::float3 dir = ray.direction;
    optix::float3 invdir = 1/dir;
    int sign[3];
    sign[0] = (invdir.x < 0);
    sign[1] = (invdir.y < 0);
    sign[2] = (invdir.z < 0);

    float tymin, tymax, tzmin, tzmax;

    optix::float3 bounds[2] = {
        optix::make_float3(0, 0, 0),
        optix::make_float3(volume_width, volume_height, volume_depth)
    };

    tmin = (bounds[sign[0]].x - orig.x) * invdir.x;
    tmax = (bounds[1-sign[0]].x - orig.x) * invdir.x;
    tymin = (bounds[sign[1]].y - orig.y) * invdir.y;
    tymax = (bounds[1-sign[1]].y - orig.y) * invdir.y;

    if ((tmin > tymax) || (tymin > tmax))
        return false;
    if (tymin > tmin)
        tmin = tymin;
    if (tymax < tmax)
        tmax = tymax;

    tzmin = (bounds[sign[2]].z - orig.z) * invdir.z;
    tzmax = (bounds[1-sign[2]].z - orig.z) * invdir.z;

    if ((tmin > tzmax) || (tzmin > tmax))
        return false;
    if (tzmin > tmin)
        tmin = tzmin;
    if (tzmax < tmax)
        tmax = tzmax;

    return true;
}

RT_PROGRAM void check_intersection(int prim_index /*There's always 1 primitive*/) {
    float tmin = 0.f, tmax = 0.f;
    bool intersected = get_aabb_ray_intersection(tmin, tmax);
    if (!intersected) {
        // rtPrintf("Nope ");
        rtThrow(RAY_MISSED_BB);
        return;
    }
    if ((tmax - tmin) < EPS) {
        return;
    }

    float n_steps = min((tmax - tmin)/stepping_distance, (float)MAX_STEPS);

    prd.r = prd.g = prd.b = 0.2;

    for(float curr_t = tmin, steps = 0; curr_t < tmax && steps < n_steps; curr_t += stepping_distance, steps++) {
        float3 point = ray.origin + curr_t*ray.direction;
        {
            int isovalue = (int) (tex3D(
                volume_texture,
                (int)(point.x + .5f),/// (float) volume_width),
                (int)(point.y + .5f),/// (float) volume_height),
                (int)(point.z + .5f )/// (float) volume_depth)
            ) * 255.99f);

            optix::float4 color = tex1D(transfer_fn_texture, isovalue) / 255.99f;
            prd.r += (1 - prd.alpha) * optix::getByIndex(color, 0)/n_steps;
            prd.g += (1 - prd.alpha) * optix::getByIndex(color, 1)/n_steps;
            prd.b += (1 - prd.alpha) * optix::getByIndex(color, 2)/n_steps;
            prd.alpha += (1 - prd.alpha) * optix::getByIndex(color, 3)/n_steps;
            if (prd.alpha > 1) {
                // break;
            }
        }
    }
}