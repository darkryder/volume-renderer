#include "hip/hip_runtime.h"
#include "device_helpers.cuh"

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(uint2,    launch_index, rtLaunchIndex, );

rtDeclareVariable(uint, volume_width, ,);
rtDeclareVariable(uint, volume_height, ,);
rtDeclareVariable(uint, volume_depth, ,);

rtDeclareVariable(float, stepping_distance, ,);

rtTextureSampler<unsigned char, 3>  volume_texture;

static __device__ bool get_aabb_ray_intersection(float &tmin, float &tmax) {
    optix::float3 orig = ray.origin;
    optix::float3 dir = ray.direction;
    optix::float3 invdir = 1/dir;
    int sign[3];
    sign[0] = (invdir.x < 0);
    sign[1] = (invdir.y < 0);
    sign[2] = (invdir.z < 0);

    float tymin, tymax, tzmin, tzmax;

    optix::float3 bounds[2] = {
        optix::make_float3(0, 0, 0),
        optix::make_float3(volume_width, volume_height, volume_depth)
    };

    tmin = (bounds[sign[0]].x - orig.x) * invdir.x;
    tmax = (bounds[1-sign[0]].x - orig.x) * invdir.x;
    tymin = (bounds[sign[1]].y - orig.y) * invdir.y;
    tymax = (bounds[1-sign[1]].y - orig.y) * invdir.y;

    if ((tmin > tymax) || (tymin > tmax))
        return false;
    if (tymin > tmin)
        tmin = tymin;
    if (tymax < tmax)
        tmax = tymax;

    tzmin = (bounds[sign[2]].z - orig.z) * invdir.z;
    tzmax = (bounds[1-sign[2]].z - orig.z) * invdir.z;

    if ((tmin > tzmax) || (tzmin > tmax))
        return false;
    if (tzmin > tmin)
        tmin = tzmin;
    if (tzmax < tmax)
        tmax = tzmax;

    return true;
}

RT_PROGRAM void check_intersection(int prim_index /*There's always 1 primitive*/) {
    float tmin = 0.f, tmax = 0.f;
    bool intersected = get_aabb_ray_intersection(tmin, tmax);
    if (!intersected) {
        rtPrintf("Nope ");
        rtThrow(RAY_MISSED_BB);
        return;
    }

    float n_steps = (tmax - tmin)/stepping_distance;

    for(float curr_t = tmin; curr_t < tmax; curr_t += stepping_distance) {
        float3 point = ray.origin + curr_t*ray.direction;
        // rtPrintf("Accessing %f %f %f\n", point.x / (float) volume_width, point.y / (float) volume_height, point.z / (float) volume_depth);
        prd.result += (tex3D(
            volume_texture,
            point.x / (float) volume_width,
            point.y / (float) volume_height,
            point.z / (float) volume_depth
        )/n_steps)/volume_depth;
    }
}