#include "hip/hip_runtime.h"
#include "device_helpers.cuh"

rtDeclareVariable(float3,   bg_colour, , );
rtDeclareVariable(uint2,    launch_index, rtLaunchIndex, );
rtBuffer<uchar4, 2>         output_buffer;

RT_PROGRAM void miss() {
    output_buffer[launch_index] = make_colour(bg_colour);
}
