#include "hip/hip_runtime.h"
#include "device_helpers.cuh"

rtDeclareVariable(float3,   exception_colour, , );
rtDeclareVariable(uint2,    launch_index, rtLaunchIndex, );
rtBuffer<uchar4, 2>         output_buffer;


RT_PROGRAM void exception() {
    rtPrintf("Exception at %d\n", launch_index);
    output_buffer[launch_index] = make_colour(exception_colour);
}
