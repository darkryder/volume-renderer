#include "hip/hip_runtime.h"
#include <optix_world.h>

rtDeclareVariable(float3, bg_color, , );

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(rtObject,      top_object, , );


RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}

static __device__ __inline__ optix::uchar4 make_color(const optix::float3& c)
{
    return optix::make_uchar4( static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* B */
                               static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
                               static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* R */
                               255u);                                                 /* A */
}

RT_PROGRAM void pinhole_camera()
{
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = optix::normalize(d.x*U + d.y*V + W);

  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}

rtDeclareVariable(float3, bad_color, , );
RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_color( bad_color );
}
